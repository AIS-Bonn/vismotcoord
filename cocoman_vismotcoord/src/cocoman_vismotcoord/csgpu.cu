#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "csgpu.h"
#include <iostream>
#include "stdio.h"

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}
__device__  unsigned int  my_rand(unsigned int *seed) {

    // constants for random no gen.
    unsigned long a = 16807;
    unsigned long m = 2147483647;   	// 2^31 - 1
    unsigned long x = (unsigned long) *seed;

    x = (a * x)%m;

    *seed = (unsigned int) x;

    return x;
}



__global__ void run(int xNr, int yNr, int zNr,int partNr,
                    float3 minPt, float3 maxPt,float gridsize,
                    float3 *cloudpos,  float3 *cloudhsv, int cloudSize,
                    float3 *partpos,  float3 *partrot, int histsize, float *refhist, float *d_weights)
{

//    int id = blockIdx.x* blockDim.x + threadIdx.x;
//    printf("ind of %i is started\n", id);

    int id =  blockIdx.x;
    int pId = threadIdx.x;

//    int id =  blockIdx.x*blockDim.x + threadIdx.x;
//    int pId = threadIdx.x;

    if (id+1>partNr)  {return;}
    float px,py,pz,roll,pitch,yaw;

    px = partpos[id].x; py = partpos[id].y; pz = partpos[id].z;
    roll = partrot[id].x; pitch = partrot[id].y; yaw = partrot[id].z;

    float t[4][4];

    float A = cosf (yaw), B = sinf (yaw), C = cosf (pitch), D = sinf (pitch),
    E = cosf (roll), F = sinf (roll), DE = D*E, DF = D*F;
    t[0][0] = A*C; t[0][1] = A*DF - B*E; t[0][2] = B*F + A*DE; t[0][3] = px;
    t[1][0] = B*C; t[1][1] = A*E + B*DF; t[1][2] = B*DE - A*F; t[1][3] = py;
    t[2][0] = -D;  t[2][1] = C*F;        t[2][2] = C*E;        t[2][3] = pz;
    t[3][0] = 0.f; t[3][1] = 0.f;        t[3][2] = 0.f;        t[3][3] = 1.f;


    //printf("time of ind %i is %i\n", id, clock());
//    int indOffset = histsize * id; int nrPerThr;
//     __shared__ float hists[10000]; indOffset=0;
//    nrPerThr = (int)ceil((float)histsize/blockDim.x);
//    for(int i=pId*nrPerThr; i<min(nrPerThr*(pId+1),histsize); i++){
//        hists[i]=0.f;
//    }
//    __syncthreads();


    int nrPerThr = (int)ceil( ((float)cloudSize) / ((float)blockDim.x) );
//    printf("number per thread is %i\n", nrPerThr);

    //    {printf("threadidx is %i start is  %i end is %i\n",pId, nrPerThr*pId, min(nrPerThr*(pId+1),cloudSize));}




    float thisCorr = 0.f;

    int thres = (int)ceil(512.f/(float)partNr * 100.f);
    unsigned int seed = blockIdx.x+threadIdx.x;
    for(int i=nrPerThr*pId; i<min(nrPerThr*(pId+1),cloudSize); i++){
//    for(int i=0; i<cloudSize; i++){

//        unsigned int rand_ = my_rand(&seed)%100;
//        if (rand_>thres){continue;}
//        if ((i+id+pId)% 5 !=0){continue;}

        float _x = cloudpos[i].x, _y = cloudpos[i].y, _z = cloudpos[i].z;

        float x = t[0][0]*_x + t[0][1]*_y + t[0][2]*_z + t[0][3];
        float y = t[1][0]*_x + t[1][1]*_y + t[1][2]*_z + t[1][3];
        float z = t[2][0]*_x + t[2][1]*_y + t[2][2]*_z + t[2][3];
        float h=cloudhsv[i].x, s=cloudhsv[i].y, v=cloudhsv[i].z;


        int grayindex;
        float weightgray, weightcolor;
        int hindex1,hindex2;
        float hweight1, hweight2;
        h = h*360.0f;
//        if(h>=0   && h <=60)  {hindex1=0; hindex2=1; hweight2= h/60.0f;          hweight1=1.0f -hweight2;}
//        if(h>60  && h <=120)  {hindex1=1; hindex2=2; hweight2= (h-60.0f)/60.0f;  hweight1=1.0f -hweight2;}
//        if(h>120 && h <=180)  {hindex1=2; hindex2=3; hweight2= (h-120.0f)/60.0f; hweight1=1.0f -hweight2;}
//        if(h>180 && h <=240)  {hindex1=3; hindex2=4; hweight2= (h-180.0f)/60.0f; hweight1=1.0f -hweight2;}
//        if(h>240 && h <=300)  {hindex1=4; hindex2=5; hweight2= (h-240.0f)/60.0f; hweight1=1.0f -hweight2;}
//        if(h>300 && h <=360)  {hindex1=5; hindex2=0; hweight2= (h-300.0f)/60.0f; hweight1=1.0f -hweight2;}

        hindex1 = (int)floor(h/60.f);
        if (hindex1==5){
            hindex2=0;
        }else{
            hindex2=hindex1+1;
        }

        hweight2 = (h- 60.f*(float)hindex1)/60.f; hweight1 = 1.f - hweight2;

        if(v<0.5) {grayindex = 6;}
        if(v>=0.5){grayindex = 7;}
        weightcolor = pow(s,(0.14f * pow(1.0f/v,0.9f)));
        weightgray  = 1.0f - weightcolor;
        if( v<0.1f){weightcolor=0.0f; weightgray=1.0f;}

//        weightcolor=0.f; weightgray=1.f; grayindex=7;

        hweight1 = weightcolor * hweight1;
        hweight2 = weightcolor * hweight2;

        int xInd, yInd, zInd, theInd;
        xInd = (int)floorf( (x - minPt.x)/gridsize );
        if((xInd >= xNr) || (xInd<0)) {continue;}
        yInd = (int)floorf( (y - minPt.y)/gridsize );
        if((yInd >= yNr) || (yInd<0)) {continue;}
        zInd = (int)floorf( (z - minPt.z)/gridsize );
        if((zInd >= zNr) || (zInd<0)) {continue;}
        theInd = zInd*xNr*yNr + xInd*yNr + yInd;

//        printf("indexs are %i %i %i\n", xInd, yInd, zInd);
//        x = x - gridsize*(float)xInd - minPt.x;
//        y = y - gridsize*(float)yInd - minPt.y;
//        z = z - gridsize*(float)zInd - minPt.z;
//        int xPlusInd = zInd*xNr*yNr + (xInd+1)*yNr + yInd;
//        int yPlusInd = zInd*xNr*yNr + xInd*yNr + (yInd+1);
//        int zPlusInd = (zInd+1)*xNr*yNr + xInd*yNr + yInd;
//        float weight1, weight2;
//        float incrValue =1.f;

//        weight2 = x/gridsize; weight1 = 1.f-weight2;  //weight2 *= weight2; weight1*=weight1;
//        thisCorr += (refhist[hindex1 + 24*theInd + 0] *  hweight1*weight1*incrValue);
//        thisCorr += (refhist[hindex1 + 24*xPlusInd + 0] *   hweight1*weight2*incrValue);
//        thisCorr += (refhist[hindex2 + 24*theInd + 0]   *   hweight2*weight1*incrValue);
//        thisCorr += (refhist[hindex2 + 24*xPlusInd + 0]  *   hweight2*weight2*incrValue);
//        thisCorr += (refhist[grayindex + 24*theInd + 0]  *   weightgray*weight1*incrValue);
//        thisCorr += (refhist[grayindex + 24*xPlusInd + 0] *   weightgray*weight2*incrValue);

//        weight2 = y/gridsize; weight1 = 1.f-weight2;  //weight2 *= weight2; weight1*=weight1;
//        thisCorr += (refhist[hindex1 + 24*theInd + 8] *   hweight1*weight1*incrValue);
//        thisCorr += (refhist[hindex1 + 24*yPlusInd + 8] *   hweight1*weight2*incrValue);
//        thisCorr += (refhist[hindex2 + 24*theInd + 8]   *   hweight2*weight1*incrValue);
//        thisCorr += (refhist[hindex2 + 24*yPlusInd + 8]  *   hweight2*weight2*incrValue);
//        thisCorr += (refhist[grayindex + 24*theInd + 8]  *   weightgray*weight1*incrValue);
//        thisCorr += (refhist[grayindex + 24*yPlusInd + 8] *   weightgray*weight2*incrValue);

//        weight2 = z/gridsize; weight1 = 1.f-weight2; // weight2 *= weight2; weight1*=weight1;
//        thisCorr += (refhist[hindex1 + 24*theInd + 16 ] *  hweight1*weight1*incrValue);
//        thisCorr += (refhist[hindex1 + 24*zPlusInd + 16 ] *   hweight1*weight2*incrValue);
//        thisCorr += (refhist[hindex2 + 24*theInd + 16 ]  *   hweight2*weight1*incrValue);
//        thisCorr += (refhist[hindex2 + 24*zPlusInd + 16 ]  *   hweight2*weight2*incrValue);
//        thisCorr += (refhist[grayindex + 24*theInd + 16 ]  *   weightgray*weight1*incrValue);
//        thisCorr += (refhist[grayindex + 24*zPlusInd + 16 ] *   weightgray*weight2*incrValue);


        x = x - gridsize*(float)xInd - minPt.x;
        y = y - gridsize*(float)yInd - minPt.y;
        z = z - gridsize*(float)zInd - minPt.z;

//        float numsafe=1000.f;
//        x *= numsafe; y*=numsafe; z*=numsafe; gridsize*=numsafe ;

        float x_=gridsize-x,  y_=gridsize-y,  z_=gridsize-z;
        float dist0 = sqrt(x*x + y*y +z*z);
        int xPInd = zInd*xNr*yNr + (xInd+1)*yNr + yInd;  float distxP = sqrt(x_*x_ + y*y +z*z) ;
        int xyPInd = zInd*xNr*yNr + (xInd+1)*yNr + yInd+1; float distxyP = sqrt(x_*x_ + y_*y_ +z*z);
        int xzPInd = (zInd+1)*xNr*yNr + (xInd+1)*yNr + yInd; float distxzP = sqrt(x_*x_ + y*y +z_*z_);
        int xyzPInd = (zInd+1)*xNr*yNr + (xInd+1)*yNr + yInd+1; float distxyzP = sqrt(x_*x_ + y_*y_ +z_*z_);
        int yPInd = zInd*xNr*yNr + xInd*yNr + (yInd+1); float distyP = sqrt(x*x + y_*y_ +z*z);
        int yzPInd = (zInd+1)*xNr*yNr + xInd*yNr + (yInd+1); float distyzP = sqrt(x*x + y_*y_ +z_*z_);
        int zPInd = (zInd+1)*xNr*yNr + xInd*yNr + yInd;  float distzP = sqrt(x*x + y*y +z_*z_);

        float incrValue =1.f; float weight; int offset;
        float bandwidth = gridsize;
        float gridsize3= 0.000001f;//gridsize*gridsize*gridsize;
//        weight = max(0.f, (bandwidth-dist0))/gridsize;
        weight = max(0.f, ((x_*y_*z_)/gridsize3));
        offset = 8*theInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxP))/gridsize;
        weight = max(0.f, ((x*y_*z_)/gridsize3));
        offset = 8*xPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxyP))/gridsize;
        weight = max(0.f, ((x*y*z_)/gridsize3));
        offset = 8*xyPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxzP))/gridsize;
        weight = max(0.f, ((x*y_*z)/gridsize3));
        offset = 8*xzPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distxyzP))/gridsize;
        weight = max(0.f, ((x*y*z)/gridsize3));
        offset = 8*xyzPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distyP))/gridsize;
        weight = max(0.f, ((x_*y*z_)/gridsize3));
        offset = 8*yPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distyzP))/gridsize;
        weight = max(0.f, ((x_*y*z)/gridsize3));
        offset = 8*yzPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);

//        weight = max(0.f, (bandwidth-distzP))/gridsize;
        weight = max(0.f, ((x_*y_*z)/gridsize3));
        offset = 8*zPInd;
        thisCorr += (refhist[hindex1 + offset] * hweight1*weight);
        thisCorr += (refhist[hindex2 + offset] * hweight2*weight);
        thisCorr += (refhist[grayindex + offset] * weightgray*weight);
    }

    atomicAdd(&d_weights[id], thisCorr);


//    nrPerThr = (int)ceil((float)histsize/blockDim.x);
//    float thisdiff=0.f;
//    for(int i=pId*nrPerThr; i<min(nrPerThr*(pId+1),histsize); i++){
////            d_weights[id]+= pow( (fabsf(hists[i + indOffset] - refhist[i])),2);
//        thisdiff += powf( (fabsf(hists[i + indOffset] - refhist[i])),1);
//    }
//    atomicAdd(&d_weights[id], thisdiff);

//    __syncthreads();

}

std::vector<float> CSGPU::compute()
{
    float *d_weights;
    float *weights;
    weights = (float *)malloc(partNr * sizeof(float));


//    size_t histSize = (xNr+1) * (yNr+1) * (zNr+1) * 24;
    size_t histSize = histsize;

    size_t histSize_ = partNr*histsize;
    float* d_hists;

    hipMalloc((void **) &d_weights, partNr*sizeof(float));
    hipMemset(d_weights, 0.f, partNr*sizeof(float));

//    hipMalloc((void **) &d_hists, histSize_*sizeof(float));
//    hipMemset(d_hists, 0.f, histSize_*sizeof(float));

//    hipMemcpy(d_hists, hists, histSize_*sizeof(float), hipMemcpyHostToDevice);
//    hipMemset(d_weights, 0, partNr*sizeof(float));

    float3 *d_cloudpos, *d_cloudhsv, *d_partpos, *d_partrot, *d_refcloud;
    float *d_refhist;
    hipMalloc((void **) &d_cloudpos, cloudSize*sizeof(float3));
    hipMalloc((void **) &d_cloudhsv, cloudSize*sizeof(float3));
    hipMalloc((void **) &d_partpos, partNr*sizeof(float3));
    hipMalloc((void **) &d_partrot, partNr*sizeof(float3));
    hipMalloc((void **) &d_refhist, histSize*sizeof(float));



    hipMemcpy(d_cloudpos, cloudpos, cloudSize*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_cloudhsv, cloudhsv, cloudSize*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_partpos, partpos, partNr*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_partrot, partrot, partNr*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_refhist, refhist, histSize*sizeof(float), hipMemcpyHostToDevice);



    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // Do calculation on device:

    int threadDim = std::min( (int)floor(65535.f/(float)partNr),  512);
    if (threadDim>32){
      threadDim = threadDim - threadDim%32;
    }

     run<<<partNr,512>>>(xNr, yNr, zNr, partNr,
                     minPt, maxPt, gridsize,
                     d_cloudpos,  d_cloudhsv, cloudSize,
                     d_partpos, d_partrot, histSize, d_refhist, d_weights);

//    run<<<100,100>>>(xNr, yNr, zNr, partNr,
//                    minPt, maxPt, gridsize,
//                    d_cloudpos,  d_cloudhsv, cloudSize,
//                    d_partpos, d_partrot, histSize, d_refhist, d_weights);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);

    hipMemcpy(weights, d_weights, partNr*sizeof(float), hipMemcpyDeviceToHost);

    float *hists;
//    hists = (float *)malloc(histSize_ * sizeof(float));
//    hipMemcpy(hists, d_hists, histSize_*sizeof(float), hipMemcpyDeviceToHost);


//    for(int i=0; i<100; i++){
//        std::cout<<weights[i]<<std::endl;
//    }
    hipFree(d_weights); hipFree(d_cloudpos);   hipFree(d_cloudhsv);
    hipFree(d_partpos); hipFree(d_partrot);    hipFree(d_refhist); hipFree(d_hists);
////    for(int i=0; i<partNr; i++){
////        std::cout<<i<<"th weight is: "<<weights[i]<<std::endl;
//////        printf ("%i th Decimals: %d \n", i, weights[i]);
////    }
    std::vector<float> out; out.resize(partNr);
    for (size_t i=0; i<partNr; i++){
        out[i] = weights[i];
    }

    free(weights); free(cloudpos); free(cloudhsv);
    free(partpos); free(partrot); free(refhist); free(hists);

    return out;

//    int *dev_c;
//    hipMalloc((void**)&dev_c, sizeof(int));
//    add<<<1,1>>>(a,b,dev_c);
//    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
////    std::cout<<a<<"+"<<b" = "<< c<<std::endl;
//    hipFree(dev_c);
}

