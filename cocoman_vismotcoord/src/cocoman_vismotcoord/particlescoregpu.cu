#include "hip/hip_runtime.h"
#include "particlescoregpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "stdio.h"
using namespace std;

ParticleScoreGPU::ParticleScoreGPU(int numParticles_, int numSceneP_, int numModelF_, float gridSize_){
    numParticles = numParticles_;
    numSceneP = numSceneP_;
    numModelF = numModelF_;
    gridSize = gridSize_;

    scenePos = (float3 *)malloc(numSceneP * sizeof(float3));
    sceneNorm = (float3 *)malloc(numSceneP * sizeof(float3));
    modelVertPos = (float3 *)malloc(numModelF * 3 * sizeof(float3));
    modelVertNorm = (float3 *)malloc(numModelF * 3 * sizeof(float3));

    particlePos = (float3 *)malloc(numParticles * sizeof(float3));
    particleRPY = (float3 *)malloc(numParticles * sizeof(float3));

    weights = (float *)malloc(numParticles * sizeof(float));
    nCrsp = (int *)malloc(numParticles * sizeof(int));
    //    crspdFacesPos = (float3 *)malloc(numSceneP * sizeof(float3));

    minBound = (float3 *)malloc(sizeof(float3));
    maxBound = (float3 *)malloc(sizeof(float3));


}

ParticleScoreGPU::~ParticleScoreGPU()
{

}


__global__ void run(int numParticles, int numSceneP, int numModelF,
                    float3 *minBound, float3 *maxBound, float gridSize, int numGridX, int numGridY, int numGridZ,
                    float3 *scenePos, float3 *sceneNorm, float3 *modelVertPos,
                    float3 *particlePos, float3 *particleRPY, float3 *crspdFacesPos, float *weights, int *nCrsp,
                    float3 *modelGridMapPos, float3 *modelGridMapNorm, int *modelGridMapNVert)
{
    int n =  blockIdx.x;
    int pId = threadIdx.x;
    if (n+1>numParticles)  {return;}

    // transformation matrix
    float px = particlePos[n].x;
    float py = particlePos[n].y;
    float pz = particlePos[n].z;
    float roll = particleRPY[n].x;
    float pitch = particleRPY[n].y;
    float yaw = particleRPY[n].z;
    float t[4][4];

    float A = cosf (yaw), B = sinf (yaw), C = cosf (pitch), D = sinf (pitch),
            E = cosf (roll), F = sinf (roll), DE = D*E, DF = D*F;
    t[0][0] = A*C; t[0][1] = A*DF - B*E; t[0][2] = B*F + A*DE; t[0][3] = px;
    t[1][0] = B*C; t[1][1] = A*E + B*DF; t[1][2] = B*DE - A*F; t[1][3] = py;
    t[2][0] = -D;  t[2][1] = C*F;        t[2][2] = C*E;        t[2][3] = pz;
    t[3][0] = 0.f; t[3][1] = 0.f;        t[3][2] = 0.f;        t[3][3] = 1.f;

    float likelihood = 0.;
    int nrPerThr = (int)ceil( ((float)numSceneP) / ((float)blockDim.x) );
    // for each scene point
    for(int i=nrPerThr*pId;i<min(nrPerThr*(pId+1),numSceneP);i++){
        float3 sceneP = scenePos[i];
        float3 sceneN = sceneNorm[i];

        // transform pos and norm
        float3 transPos, transNorm;
        transPos.x = t[0][0]*sceneP.x + t[0][1]*sceneP.y + t[0][2]*sceneP.z + t[0][3];
        transPos.y = t[1][0]*sceneP.x + t[1][1]*sceneP.y + t[1][2]*sceneP.z + t[1][3];
        transPos.z = t[2][0]*sceneP.x + t[2][1]*sceneP.y + t[2][2]*sceneP.z + t[2][3];
        transNorm.x = t[0][0]*sceneN.x + t[0][1]*sceneN.y + t[0][2]*sceneN.z + t[0][3];
        transNorm.y = t[1][0]*sceneN.x + t[1][1]*sceneN.y + t[1][2]*sceneN.z + t[1][3];
        transNorm.z = t[2][0]*sceneN.x + t[2][1]*sceneN.y + t[2][2]*sceneN.z + t[2][3];

        //        transPos = sceneP;
        /*
        // find the closest face and show the correspondence?
        float min = 0.;
        int argmin = 0;
        for(int j=0;j<numModelF;j++){
            float dist = 0.;
            for(int k=0;k<3;k++){
                dist += sqrt(pow((transPos.x - modelVertPos[j*3+k].x),2)
                        + pow((transPos.y - modelVertPos[j*3+k].y),2)
                        + pow((transPos.z - modelVertPos[j*3+k].z),2));
            }
            dist /= 3.;
            if(j == 0){
                min = dist;
                argmin = j;
            }
            else{
                if(dist < min){
                    min = dist;
                    argmin = j;
                }
            }
        }
        //            cout<<argmin<<" ";
        crspdFacesPos[i].x = (modelVertPos[argmin*3+0].x + modelVertPos[argmin*3+1].x + modelVertPos[argmin*3+2].x) / 3.;
        crspdFacesPos[i].y = (modelVertPos[argmin*3+0].y + modelVertPos[argmin*3+1].y + modelVertPos[argmin*3+2].y) / 3.;
        crspdFacesPos[i].z = (modelVertPos[argmin*3+0].z + modelVertPos[argmin*3+1].z + modelVertPos[argmin*3+2].z) / 3.;
*/
        float likelihood_point = 0.;
        float dist_error;
        // find index of transPos in the gridmap
        int idx_x = (transPos.x-minBound->x)/gridSize;
        int idx_y = (transPos.y-minBound->y)/gridSize;
        int idx_z = (transPos.z-minBound->z)/gridSize;

        if(idx_x < 0 || idx_x >= numGridX || idx_y < 0 || idx_y >= numGridY ||idx_z < 0 || idx_z >= numGridZ){
            //            crspdFacesPos[i].x = 0;
            //            crspdFacesPos[i].y = 0;
            //            crspdFacesPos[i].z = 0;
            dist_error = GRIDSIZE;
        }
        else{
            int idx = (idx_x * numGridY + idx_y) * numGridZ + idx_z;
            if(modelGridMapNVert[idx] != 0){
                atomicAdd(&nCrsp[n], 1);
                //                crspdFacesPos[i].x = modelGridMapPos[idx].x;
                //                crspdFacesPos[i].y = modelGridMapPos[idx].y;
                //                crspdFacesPos[i].z = modelGridMapPos[idx].z;
                if(!USENORM)
                    dist_error = pow(transPos.x-modelGridMapPos[idx].x,2) + pow(transPos.y-modelGridMapPos[idx].y,2) + pow(transPos.z-modelGridMapPos[idx].z,2);
//                        dist_error = 0;
                else
                    dist_error = pow((transPos.x-modelGridMapPos[idx].x)*modelGridMapNorm[idx].x + (transPos.y-modelGridMapPos[idx].y)*modelGridMapNorm[idx].y + (transPos.z-modelGridMapPos[idx].z)*modelGridMapNorm[idx].z,2);
//                    dist_error = 0;
                //                float norm1 = sqrt(pow(modelGridMapNorm[idx].x,2)+pow(modelGridMapNorm[idx].y,2)+pow(modelGridMapNorm[idx].z,2));
                //                float norm2 = sqrt(pow(transNorm.x,2)+pow(transNorm.y,2)+pow(transNorm.z,2));
                //                float dot = modelGridMapNorm[idx].x*transNorm.x + modelGridMapNorm[idx].y*transNorm.y + modelGridMapNorm[idx].z*transNorm.z;
                //                dist_angle = acos(dot/(norm1*norm2)) / 3.141592;

            }
            else{
                //                crspdFacesPos[i].x = 0;
                //                crspdFacesPos[i].y = 0;
                //                crspdFacesPos[i].z = 0;
                dist_error = GRIDSIZE;
            }
        }
        likelihood_point = (1. * 1 * dist_error);
//        if(USENORM) likelihood_point += (1. * 1 * dist_angle);
        likelihood += likelihood_point;
    }

    // calculate weight
    atomicAdd(&weights[n], likelihood);

}

std::vector<float> ParticleScoreGPU::compute(float3 *crspdFacesPos)
{
    numGridX = (maxBound->x - minBound->x) / gridSize;
    numGridY = (maxBound->y - minBound->y) / gridSize;
    numGridZ = (maxBound->z - minBound->z) / gridSize;

    modelGridMapPos = (float3 *)malloc(numGridX*numGridY*numGridZ * sizeof(float3));
    modelGridMapNorm = (float3 *)malloc(numGridX*numGridY*numGridZ * sizeof(float3));
    modelGridMapNVert = (int *)malloc(numGridX*numGridY*numGridZ * sizeof(int));
    memset(modelGridMapPos, 0.f, numGridX*numGridY*numGridZ*sizeof(float3));
    memset(modelGridMapNorm, 0.f, numGridX*numGridY*numGridZ*sizeof(float3));
    memset(modelGridMapNVert, 0, numGridX*numGridY*numGridZ*sizeof(int));

    // memory set on the device
    float *d_weights;
    hipMalloc((void **) &d_weights, numParticles*sizeof(float));
    hipMemset(d_weights, 0.f, numParticles*sizeof(float));

    int *d_nCrsp;
    hipMalloc((void **) &d_nCrsp, numParticles*sizeof(int));
    hipMemset(d_nCrsp, 0, numParticles*sizeof(int));

    float3 *d_crspdFacesPos;
    hipMalloc((void **) &d_crspdFacesPos, numSceneP*sizeof(float3));
    hipMemset(d_crspdFacesPos, 0.f, numSceneP*sizeof(float3));

    // input
    float3 *d_scenePos, *d_sceneNorm, *d_modelVertPos, *d_particlePos, *d_particleRPY, *d_modelGridMapPos, *d_modelGridMapNorm;
    float3 *d_minBound, *d_maxBound;

    int *d_modelGridMapNVert;
    hipMalloc((void **) &d_scenePos, numSceneP*sizeof(float3));
    hipMalloc((void **) &d_sceneNorm, numSceneP*sizeof(float3));
    hipMalloc((void **) &d_modelVertPos, numModelF * 3 *sizeof(float3));
    hipMalloc((void **) &d_particlePos, numParticles*sizeof(float3));
    hipMalloc((void **) &d_particleRPY, numParticles*sizeof(float3));
    hipMalloc((void **) &d_modelGridMapPos, numGridX*numGridY*numGridZ*sizeof(float3));
    hipMalloc((void **) &d_modelGridMapNorm, numGridX*numGridY*numGridZ*sizeof(float3));
    hipMalloc((void **) &d_modelGridMapNVert, numGridX*numGridY*numGridZ*sizeof(int));
    hipMalloc((void **) &d_minBound, sizeof(float3));
    hipMalloc((void **) &d_maxBound, sizeof(float3));

    hipMemset(d_modelGridMapPos, 0.f, numGridX*numGridY*numGridZ*sizeof(float3));
    hipMemset(d_modelGridMapNorm, 0.f, numGridX*numGridY*numGridZ*sizeof(float3));
    hipMemset(d_modelGridMapNVert, 0, numGridX*numGridY*numGridZ*sizeof(int));

    // compute grid space
    //    modelGridMapNVert[3200688] ++;
//    cout<<"# gridsize: "<<numGridX*numGridY*numGridZ<<endl;

    // for each face
    for(int i=0;i<numModelF;i++){
        // make a boundary
        float3 min = modelVertPos[i*3+0];
        float3 max = modelVertPos[i*3+0];
        for(int k=1;k<3;k++){
            if(modelVertPos[i*3+k].x < min.x) min.x = modelVertPos[i*3+k].x;
            if(modelVertPos[i*3+k].x > max.x) max.x = modelVertPos[i*3+k].x;
            if(modelVertPos[i*3+k].y < min.y) min.y = modelVertPos[i*3+k].y;
            if(modelVertPos[i*3+k].y > max.y) max.y = modelVertPos[i*3+k].y;
            if(modelVertPos[i*3+k].z < min.z) min.z = modelVertPos[i*3+k].z;
            if(modelVertPos[i*3+k].z > max.z) max.z = modelVertPos[i*3+k].z;
        }
        //convert minmax to idxes of grid
        int idx_min_x = (min.x-minBound->x)/gridSize;
        int idx_max_x = (max.x-minBound->x)/gridSize;
        int idx_min_y = (min.y-minBound->y)/gridSize;
        int idx_max_y = (max.y-minBound->y)/gridSize;
        int idx_min_z = (min.z-minBound->z)/gridSize;
        int idx_max_z = (max.z-minBound->z)/gridSize;

        //        cout<<idx_min_x<<" "<<idx_max_x<<" "<<idx_min_y<<" "<<idx_max_y<<" "<<idx_min_z<<" "<<idx_max_z<<endl;

        if(idx_max_x >= numGridX) idx_max_x = numGridX-1;
        if(idx_max_y >= numGridY) idx_max_y = numGridY-1;
        if(idx_max_z >= numGridZ) idx_max_z = numGridZ-1;
        if(idx_min_x < 0) idx_min_x = 0;
        if(idx_min_y < 0) idx_min_y = 0;
        if(idx_min_z < 0) idx_min_z = 0;

        // find a grid to fill
        for(int idx_x = idx_min_x; idx_x <= idx_max_x; idx_x ++)
            for(int idx_y = idx_min_y; idx_y <= idx_max_y; idx_y ++)
                for(int idx_z = idx_min_z; idx_z <= idx_max_z; idx_z ++){
                    int idx = (idx_x * numGridY + idx_y) * numGridZ + idx_z;
                    if(idx >= numGridX*numGridY*numGridZ) cout<<"!!!!!!!!!!!!!!!!!!!!!";

                    // find the closest vertex of three
                    float3 gridPos;
                    gridPos.x = minBound->x + idx_x * gridSize + gridSize/2.;
                    gridPos.y = minBound->y + idx_y * gridSize + gridSize/2.;
                    gridPos.z = minBound->z + idx_z * gridSize + gridSize/2.;

                    float distMin = sqrt(pow(gridPos.x-modelVertPos[i*3+0].x,2)+pow(gridPos.y-modelVertPos[i*3+0].y,2)+pow(gridPos.z-modelVertPos[i*3+0].z,2));
                    int minVert = 0;

                    for(int k=1;k<3;k++){
                        float dist = sqrt(pow(gridPos.x-modelVertPos[i*3+k].x,2)+pow(gridPos.y-modelVertPos[i*3+k].y,2)+pow(gridPos.z-modelVertPos[i*3+k].z,2));
                        if(dist < distMin) {
                            distMin = dist;
                            minVert = k;
                        }
                    }
                    //                    cout<<idx<<" ";
                    if(modelGridMapNVert[idx] == 0){
                        modelGridMapPos[idx] = modelVertPos[i*3+minVert];
                        if(USENORM) modelGridMapNorm[idx] = modelVertNorm[i*3+minVert];
                    }
                    else{
                        modelGridMapPos[idx].x = (modelGridMapPos[idx].x*modelGridMapNVert[idx] + modelVertPos[i*3+minVert].x) / (modelGridMapNVert[idx]+1.);
                        modelGridMapPos[idx].y = (modelGridMapPos[idx].y*modelGridMapNVert[idx] + modelVertPos[i*3+minVert].y) / (modelGridMapNVert[idx]+1.);
                        modelGridMapPos[idx].z = (modelGridMapPos[idx].z*modelGridMapNVert[idx] + modelVertPos[i*3+minVert].z) / (modelGridMapNVert[idx]+1.);

                        if(USENORM){
                            modelGridMapNorm[idx].x = (modelGridMapNorm[idx].x*modelGridMapNVert[idx] + modelVertNorm[i*3+minVert].x) / (modelGridMapNVert[idx]+1.);
                            modelGridMapNorm[idx].y = (modelGridMapNorm[idx].y*modelGridMapNVert[idx] + modelVertNorm[i*3+minVert].y) / (modelGridMapNVert[idx]+1.);
                            modelGridMapNorm[idx].z = (modelGridMapNorm[idx].z*modelGridMapNVert[idx] + modelVertNorm[i*3+minVert].z) / (modelGridMapNVert[idx]+1.);
                        }
                    }

                    modelGridMapNVert[idx] ++;

                }

        // for each grid
        // update normalvec and # of vert for the selected grid

    }

    // memory copy from host to device
    hipMemcpy(d_scenePos, scenePos, numSceneP*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_sceneNorm, sceneNorm, numSceneP*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_modelVertPos, modelVertPos, numModelF * 3 *sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_particlePos, particlePos, numParticles*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_particleRPY, particleRPY, numParticles*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_modelGridMapPos, modelGridMapPos, numGridX*numGridY*numGridZ*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_modelGridMapNorm, modelGridMapNorm, numGridX*numGridY*numGridZ*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_modelGridMapNVert, modelGridMapNVert, numGridX*numGridY*numGridZ*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minBound, minBound, sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_maxBound, maxBound, sizeof(float3), hipMemcpyHostToDevice);


    // computation time
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /// computation on gpu
//    cout<<minBound->x<<" "<<minBound->y<<" "<<minBound->z<<endl;
    run<<<numParticles, 1024>>>(numParticles, numSceneP, numModelF, d_minBound, d_maxBound, gridSize, numGridX, numGridY, numGridZ,
                                d_scenePos, d_sceneNorm, d_modelVertPos, d_particlePos, d_particleRPY, d_crspdFacesPos, d_weights, d_nCrsp,
                                d_modelGridMapPos, d_modelGridMapNorm, d_modelGridMapNVert);

    /// computation on cpu

    //    // for each particle
    //    for(int n=0;n<numParticles;n++){
    //        cout<<"# particles: "<<n<<endl;
    //        // transformation matrix
    //        float px = particlePos[n].x;
    //        float py = particlePos[n].y;
    //        float pz = particlePos[n].z;
    //        float roll = particleRPY[n].x;
    //        float pitch = particleRPY[n].y;
    //        float yaw = particleRPY[n].z;
    //        float t[4][4];

    //        float A = cosf (yaw), B = sinf (yaw), C = cosf (pitch), D = sinf (pitch),
    //                E = cosf (roll), F = sinf (roll), DE = D*E, DF = D*F;
    //        t[0][0] = A*C; t[0][1] = A*DF - B*E; t[0][2] = B*F + A*DE; t[0][3] = px;
    //        t[1][0] = B*C; t[1][1] = A*E + B*DF; t[1][2] = B*DE - A*F; t[1][3] = py;
    //        t[2][0] = -D;  t[2][1] = C*F;        t[2][2] = C*E;        t[2][3] = pz;
    //        t[3][0] = 0.f; t[3][1] = 0.f;        t[3][2] = 0.f;        t[3][3] = 1.f;

    //        // for each scene point
    //        for(int i=0;i<numSceneP;i++){
    //            float3 sceneP = scenePos[i];
    //            float3 sceneN = sceneNorm[i];

    //            // transform pos and norm
    //            float3 transPos, transNorm;
    //            transPos.x = t[0][0]*sceneP.x + t[0][1]*sceneP.y + t[0][2]*sceneP.z + t[0][3];
    //            transPos.y = t[1][0]*sceneP.x + t[1][1]*sceneP.y + t[1][2]*sceneP.z + t[1][3];
    //            transPos.z = t[2][0]*sceneP.x + t[2][1]*sceneP.y + t[2][2]*sceneP.z + t[2][3];
    //            transNorm.x = t[0][0]*sceneN.x + t[0][1]*sceneN.y + t[0][2]*sceneN.z + t[0][3];
    //            transNorm.y = t[1][0]*sceneN.x + t[1][1]*sceneN.y + t[1][2]*sceneN.z + t[1][3];
    //            transNorm.z = t[2][0]*sceneN.x + t[2][1]*sceneN.y + t[2][2]*sceneN.z + t[2][3];

    //            // find the closest face and show the correspondence?
    //            float min = 0.;
    //            int argmin;
    //            for(int j=0;j<numModelF;j++){
    //                float dist = 0.;
    //                for(int k=0;k<3;k++){
    //                    dist += sqrt(pow((transPos.x - modelVertPos[j*3+k].x),2)
    //                            + pow((transPos.y - modelVertPos[j*3+k].y),2)
    //                            + pow((transPos.z - modelVertPos[j*3+k].z),2));
    //                }
    //                dist /= 3.;
    //                if(j == 0){
    //                    min = dist;
    //                    argmin = j;
    //                }
    //                else{
    //                    if(dist < min){
    //                        min = dist;
    //                        argmin = j;
    //                    }
    //                }
    //            }
    //            //            cout<<argmin<<" ";
    //            crspdFacesPos[i].x = (modelVertPos[argmin*3+0].x + modelVertPos[argmin*3+1].x + modelVertPos[argmin*3+2].x) / 3.;
    //            crspdFacesPos[i].y = (modelVertPos[argmin*3+0].y + modelVertPos[argmin*3+1].y + modelVertPos[argmin*3+2].y) / 3.;
    //            crspdFacesPos[i].z = (modelVertPos[argmin*3+0].z + modelVertPos[argmin*3+1].z + modelVertPos[argmin*3+2].z) / 3.;
    //        }
    //    }


    //    // end computation
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    if(DEBUG_ALGORITHM) cout<<"Time for the kernel: "<<time<<" ms"<<endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // memory copy from device to host
    hipMemcpy(weights, d_weights, numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(nCrsp, d_nCrsp, numParticles*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(crspdFacesPos, d_crspdFacesPos, numSceneP*sizeof(float3), hipMemcpyDeviceToHost);

    std::vector<float> out;
    out.resize(numParticles);
    for (size_t i=0; i<numParticles; i++){
//        out[i] = weights[i] / (float)nCrsp[i];
           out[i] = weights[i];
    }
    if(DEBUG_ALGORITHM) cout<<"min weight: ";
    float min;
    for(int i=0;i<out.size();i++){
        if(i==0) min = out[i];
        else if(out[i] < min) min = out[i];
    }
    if(DEBUG_ALGORITHM) cout<<min<<endl;

    // release
    hipFree(d_weights);
    hipFree(d_nCrsp);
    hipFree(d_crspdFacesPos);
    hipFree(d_scenePos);
    hipFree(d_sceneNorm);
    hipFree(d_modelVertPos);
    hipFree(d_particlePos);
    hipFree(d_particleRPY);
    hipFree(d_modelGridMapPos);
    hipFree(d_modelGridMapNorm);
    hipFree(d_modelGridMapNVert);
    hipFree(d_minBound);
    hipFree(d_maxBound);

    free(scenePos);
    free(sceneNorm);
    free(modelVertPos);
    free(particlePos);
    free(particleRPY);
    free(weights);
    free(nCrsp);
    //    free(crspdFacesPos);
    free(modelGridMapPos);
    free(modelGridMapNorm);
    free(modelGridMapNVert);
    free(minBound);
    free(maxBound);

    return out;
}
